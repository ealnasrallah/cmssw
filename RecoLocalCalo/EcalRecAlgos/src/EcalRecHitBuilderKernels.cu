#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "KernelHelpers.h"

#include "CUDADataFormats/EcalRecHitSoA/interface/EcalUncalibratedRecHit_soa.h"
#include "CUDADataFormats/EcalRecHitSoA/interface/EcalRecHit_soa.h"

//
//
#include "EcalRecHitBuilderKernels.h"


namespace ecal {
  namespace rechit {
    
    __global__
    void kernel_create_ecal_rehit(
                    uint32_t const* did_eb,
                    uint32_t const* did_ee,
                    ::ecal::reco::StorageScalarType const* amplitude_eb,   // in adc counts  
                    ::ecal::reco::StorageScalarType const* amplitude_ee,   // in adc counts  
                    ::ecal::reco::StorageScalarType* energy,   // in energy [GeV]  
                    int const nchannels
         ) {
      
      
//       
//    NB: energy   "type_wrapper<reco::StorageScalarType, L>::type" most likely std::vector<float>
//       
      
      int idx = threadIdx.x + blockDim.x*blockIdx.x;
      
      if (idx < nchannels) {
        
        // simple copy
        energy[idx] = amplitude_eb[idx];
        
      }
      
    }
    
    
    
    // host version, to be called by the plugin
    void create_ecal_rehit(
                  EventInputDataGPU const& eventInputGPU,
                  EventOutputDataGPU&      eventOutputGPU,
                  //     eventDataForScratchGPU_,
                  //     conditions,
                  //     configParameters_,
                  cuda::stream_t<>& cudaStream
             ){
    
      int nchannels = 10;
      
      unsigned int totalChannels = 10; //eventInputGPU.ebUncalibRecHits.nchannels +
//       eventInputGPU.eeUncalibRecHits.nchannels;
      
      unsigned int nchannels_per_block = 32;
      unsigned int threads_1d = 10 * nchannels_per_block;
      //   unsigned int blocks_1d = threads_1d > 10*totalChannels  ? 1 : (totalChannels*10 + threads_1d - 1) / threads_1d;
      unsigned int blocks_1d = 2;
      

//       kernel_create_ecal_rehit <<< blocks_1d, threads_1d >>> (
//         eventInputGPU.ebUncalibRecHits.did,
//         eventInputGPU.eeUncalibRecHits.did,
//         eventInputGPU.ebUncalibRecHits.amplitude, 
//         eventInputGPU.eeUncalibRecHits.amplitude, 
//         eventOutputGPU.energy,
//         nchannels
//       );
      
        
      
      
      

      // 
      // kernel
      //
        
//       kernel_create_ecal_rehit <<< blocks_1d, threads_1d >>> (
//                                amplitude,
//                                energy,
//                                nchannels
//       );
      
    }
    
    
//     
// //     error: cannot convert 'ecal::type_wrapper<float, ecal::Tag::soa>::type {aka std::vector<float, std::allocator<float> >}' to 'float*' for argument '2' to 'void ecal::rechit::create_ecal_rehit(const float*, float*, int)'
//     
// //     error: cannot convert 'ecal::type_wrapper<float, ecal::Tag::soa>::type {aka std::vector<float, CUDAHostAllocator<float, 0> >}' to 'float*' for argument '2' to 'void ecal::rechit::create_ecal_rehit(const float*, float*, int)'
//     
// //     error: cannot convert 'ecal::type_wrapper<float, ecal::Tag::soa>::type {aka std::vector<float, CUDAHostAllocator<float, 0> >}' to 'const float*' for argument '2' to 'void ecal::rechit::create_ecal_rehit(const float*, const float*, int)'
//     
// //     error: cannot convert 'ecal::type_wrapper<float, ecal::Tag::soa>::type {aka std::vector<float, CUDAHostAllocator<float, 0> >}' to 'ecal::type_wrapper<float, ecal::Tag::soa>::type* {aka std::vector<float, CUDAHostAllocator<float, 0> >*}'    
// 
//  error: no operator "=" matches these operands
//  operand types are: std::vector<ecal::reco::StorageScalarType, CUDAHostAllocator<ecal::reco::StorageScalarType, 0U>> = const float
// 
//    


    
  }
  
}

